/* ===================================================================
 * cu_fastq_stats_mmap.cc
 *  computes basic sequence stats for a FASTQ dataset using the GPU
 *  These include average A,T,C,G and N content and average quality
 *  per position in the sequence
 * Written by Assen Roguev, 2018
 * =================================================================== */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cstdint>
#include <memory.h>

#include <sys/stat.h>
#include <fcntl.h>
#include <unistd.h>
#include <sys/mman.h>
#include <errno.h>
#include <err.h>

typedef struct {
    int fp;
    char* bS;
    char* bE;
    char* lS;
    char* lE;
    struct stat fs;
} mmap_line_t;
    

// compile with -DCUDA_ERROR_CHECK toturn on error checking
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    if ( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        
        exit( -1 );
    }
#endif
    return;
}

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    cudaError err = cudaGetLastError();
    if ( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", cudaGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = cudaDeviceSynchronize();
    if( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

/* pos_stats is a cuda kernel to compuye sequence stats
 * Arguments:
 *  device char* to concatenated sequence data
 *  device float* to hold the stats
 *  size_t length of sequence read
 *  size_t chunk size 
 */
__global__
void pos_stats(char* str, float* pos_stats, size_t LEN, size_t CHUNK_SZ) {
    // define shared memory array
    extern __shared__ int spos_stats[];
    // set to 0
    if (threadIdx.x == 0) {
        for (size_t i = 0; i < 6*LEN; i++) {
            spos_stats[i] = 0;
        }
    }
    __syncthreads();
    
    // get the sequence index
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    if (ix < CHUNK_SZ) {
        for (size_t i = 0; i < LEN; i++) {
            switch (str[2*LEN*ix+i]) {
                case 'A' :  case 'a':
                    atomicAdd(&spos_stats[6*i+0], 1);
                    break;
                case 'T' :  case 't':
                    atomicAdd(&spos_stats[6*i+1], 1);
                    break;
                case 'G' :  case 'g':
                    atomicAdd(&spos_stats[6*i+2], 1);
                    break;
                case 'C' :  case 'c':
                    atomicAdd(&spos_stats[6*i+3], 1);
                    break;
                case 'N' : case 'n':
                    atomicAdd(&spos_stats[6*i+4], 1);
                    break;
            }
        // per-position quality
        atomicAdd(&spos_stats[6*i+5], str[2*LEN*ix+i+LEN]);
        }
    }
    __syncthreads();
    
    // add to the global pos_stats structure
    if (threadIdx.x == 0) {
        for (size_t i = 0; i < 6*LEN; i++)
            atomicAdd(&pos_stats[i], spos_stats[i]);
    }
}

/* print_pos_stats prints the contents of the pos_stats array
 * Arguments:
 *  device float* containing the stats to be printed
 *  size_t length of the sequence read
 *  uint64_t total number of sequences
 */
void print_pos_stats(float* d_pos_stats, size_t LEN, uint64_t totalN) {
    const char charset[] = "ATGCNQ";
    
    // copy data to hist memory
    float* tmp_pos_stats = (float*)calloc(6*LEN, sizeof(float));
    CudaSafeCall( hipMemcpy(tmp_pos_stats, d_pos_stats, 6*LEN*sizeof(float), hipMemcpyDeviceToHost) );
    
    for (int b = 0; b < 6; b++) {
        printf("@%c", charset[b]);
        for (size_t i = 0; i < LEN; i++) {
            if (b < 5)
                // average nucleotide content
                printf("\t%0.4f",(float)tmp_pos_stats[6*i+b]/totalN);
            else {
                // check if q-stats are 0 at this position
                // can happen if the LEN is bigger than the actual read length
                if (tmp_pos_stats[6*i+b] == 0)
                     printf("\t%0.4f", (float)0);
                else
                    // average quality
                    printf("\t%0.4f",(float)(tmp_pos_stats[6*i+b] - 33*totalN)/totalN);
            }
        }
        printf("\n");
    }
    
    // cleanup
    free(tmp_pos_stats);
}

/* readline reads a line from mapped memory
 * a line is defined as a string terminated by
 * '\r', '\n', '\r\n' or '\n\r'
 * Arguments
 *  mmap_line_t* pointing to a structure
 */
int readline(mmap_line_t* mm) {    
    char c;
    
    // reset lS and lE
    if (mm->lS != mm->lE) {
        if ((mm->lS = ++mm->lE) >= mm->bE)
            return 0;   // eof reached
    }
    
    while (1) {
        // see if we got "\r" or "\n" here
        if (! (*mm->lE == '\r' || *mm->lE == '\n')) {
            if (++mm->lE < mm->bE)
                continue;
            else 
                return 0;  // eof reached, no newline
        }
        
        // see if we got "\r\n" or "\n\r" here
        if (1 + mm->lE < mm->bE) {
            c = *(1 + mm->lE);
            if ( (c == '\r' || c == '\n') && c != *mm->lE) { 
                ++mm->lE;
                return 1;
            }
        }
        
#ifdef DEBUG
        for (char* i = mm->lS; i < mm->lE; i++)
            printf("%c", *i);
        printf("\n");
#endif
        return 1;
    }
}


/* read_fastq attempts to read a fastq record from a file
 * Arguments:
 *  char* to hold the read data
 *  mmap_line_t* pointing to a structure
 *  size_t for sequence read length
 *  size_t offset within the target string
 */
bool read_fastq(char* str, mmap_line_t* mm, const size_t L, size_t offset) {
    ssize_t rs;
    
    while (1) {
        if (! readline(mm) ) break;
        
        // entry point, line starts with '@'
        if (*mm->lS != '@')  { printf("1\n"); continue; }   // drop this line keep going
            
        // read sequence string, check for error and empty line
        if (readline(mm) && ((rs = mm->lE - mm->lS) > 0)) {
            if (rs < L) 
                memcpy(str+2*L*offset,mm->lS,rs*sizeof(char)); 
            else
                memcpy(str+2*L*offset,mm->lS,L*sizeof(char));
        } else break;
            
        // read '+' string, line starts with '+', check for error and not starting with '+'
        if (readline(mm) && (*mm->lS != '+')) break; 
            
        // read q-string, check for error and different length from sequence
        if (readline(mm) && ((mm->lE - mm->lS)  == rs)) { 
            if (rs < L)
                memcpy(str+2*L*offset+L,mm->lS,rs*sizeof(char));
            else 
                memcpy(str+2*L*offset+L,mm->lS,L*sizeof(char));
            
            return true;    // success
        } else break;
    }
    
    // something went wrong
    return false;
}


/* main 
 * Arguments:
 *  filename
 *  sequence read length
 *  (optional) max number of sequences to process
 * */
int main(int argc, char** argv) {
    // change this if needed (and change the kernel launch parameters below)
    size_t CHUNK_SZ = 1000000;
    
    if (argc < 3) {
        fprintf(stderr, "%s: Insufficient arguments\n", argv[0]);
        exit(1);
    }
    
    char* fname = argv[1];              // input filename
    size_t LEN = (size_t)atoi(argv[2]); // desired sequence length
    uint64_t maxSeq = 0;                // maximum number of sequences to process
    uint64_t totalN = 0;                // total number of sequences
    
    mmap_line_t mm;
    
    if (argc == 4) {
        maxSeq = (uint64_t)atoi(argv[3]);
    }
    
    // open file for reading
    mm.fp = open(fname, O_RDONLY);
    if (mm.fp == -1) { 
        err(1, "open: %s", fname);
        exit(2);
    }
 
    // populate stat structure
    if (fstat(mm.fp, &mm.fs) == -1) {
        err(1, "stat: %s", fname);
        exit(2);
        }
 
    // mmap file
    mm.bS = (char*)mmap(0, mm.fs.st_size, PROT_READ, MAP_SHARED, mm.fp, 0);
    if (mm.bS == (void*) -1) {
        err(1, "mmap: %s", fname);
        close(mm.fp);
        exit(3);
        }
        
    mm.bE = mm.bS + mm.fs.st_size;
    mm.lS = mm.lE = mm.bS;
    
    // allocate host and device memory
    char* h_str = (char*)calloc(2*LEN*CHUNK_SZ,sizeof(char));
    
    char* d_str;       // sequence
    CudaSafeCall( hipMalloc((void**)&d_str, 2*CHUNK_SZ*LEN*sizeof(char)) );
    CudaSafeCall( hipMemset(d_str, 0, 2*CHUNK_SZ*LEN*sizeof(char)) );

    float* d_pos_stats;  // pos stats    
    CudaSafeCall( hipMalloc((void**)&d_pos_stats, 6*LEN*sizeof(float)) );
    CudaSafeCall( hipMemset(d_pos_stats, 0, 6*LEN*sizeof(float)) );
    
    // main loop to go arbitrary number of sequences
    bool done = false;
    while(!done) {
        for (size_t i = 0; i < (size_t)CHUNK_SZ; i++) {
            if (read_fastq(h_str, &mm, LEN, i)) {
                totalN++;
                if (totalN == maxSeq) {
                    done = true;
                    break;
                }
            
            } else { 
                done = true; 
                break;
            }
        }

        CudaSafeCall( hipMemcpy(d_str, h_str, 2*CHUNK_SZ*LEN*sizeof(char), hipMemcpyHostToDevice) );
    
        pos_stats<<<1024,1024,6*LEN*sizeof(float)>>>(d_str, d_pos_stats, LEN, CHUNK_SZ);
        CudaCheckError();
            
        // reset arrays
        memset(h_str, 0, 2*CHUNK_SZ*LEN*sizeof(char));
        CudaSafeCall( hipMemset(d_str, 0, 2*CHUNK_SZ*LEN*sizeof(char)) );
    }
    fprintf(stderr, "\n");
    
    // close file
    munmap(mm.bS, mm.fs.st_size);
    close(mm.fp);
    
    printf("Total seqs processed: %lu\tLength: %zu\tChunk: %zu\n", totalN, LEN, CHUNK_SZ);
    print_pos_stats(d_pos_stats, LEN, totalN);
    hipDeviceSynchronize();    
    
    // cleanuup
    // device
    hipFree(d_str);
    hipFree(d_pos_stats);
    
    // host
    free(h_str);
    
    return 0;
}