/* ===================================================================
 * cu_fastq_stats.cc
 *  computes basic sequence stats for a FASTQ dataset using the GPU
 *  These include average A,T,C,G and N content and average quality
 *  per position in the sequence
 * Written by Assen Roguev, 2018
 * =================================================================== */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cstdlib>
#include <cstdint>
#include <memory.h>
#include <errno.h>
#include <err.h>

// compile with -DCUDA_ERROR_CHECK toturn on error checking
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    if ( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        
        exit( -1 );
    }
#endif
    return;
}

inline void __cudaCheckError( const char *file, const int line ) {
#ifdef CUDA_ERROR_CHECK
    cudaError err = cudaGetLastError();
    if ( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", cudaGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = cudaDeviceSynchronize();
    if( cudaSuccess != err ) {
        fprintf( stderr, "%s:%i : %s\n", file, line, cudaGetErrorString( err ) );
        exit( -1 );
    }
#endif
    return;
}

/* pos_stats is a cuda kernel to compuye sequence stats
 * Arguments:
 *  device char* to concatenated sequence data
 *  device float* to hold the stats
 *  size_t length of sequence read
 *  size_t chunk size 
 */
__global__
void pos_stats(char* str, float* pos_stats, size_t LEN, size_t CHUNK_SZ) {
    // define shared memory array
    extern __shared__ int spos_stats[];
    // set to 0
    if (threadIdx.x == 0) {
        for (size_t i = 0; i < 6*LEN; i++)
            spos_stats[i] = 0;
    }
    __syncthreads();
    
    // get the sequence index
    unsigned int ix = blockDim.x * blockIdx.x + threadIdx.x;
    if (ix < CHUNK_SZ) {
        for (size_t i = 0; i < LEN; i++) {
            switch (str[2*LEN*ix+i]) {
                case 'A' :  case 'a':
                    atomicAdd(&spos_stats[6*i+0], 1);
                    break;
                case 'T' :  case 't':
                    atomicAdd(&spos_stats[6*i+1], 1);
                    break;
                case 'G' :  case 'g':
                    atomicAdd(&spos_stats[6*i+2], 1);
                    break;
                case 'C' :  case 'c':
                    atomicAdd(&spos_stats[6*i+3], 1);
                    break;
                case 'N' : case 'n':
                    atomicAdd(&spos_stats[6*i+4], 1);
                    break;
            }
        // per-position quality
        atomicAdd(&spos_stats[6*i+5], str[2*LEN*ix+i+LEN]);
        }
    }
    __syncthreads();
    
    // add to the global pos_stats structure
    if (threadIdx.x == 0) {
        for (size_t i = 0; i < 6*LEN; i++)
            atomicAdd(&pos_stats[i], spos_stats[i]);
    }
}

/* print_pos_stats prints the contents of the pos_stats array
 * Arguments:
 *  device float* containing the stats to be printed
 *  size_t length of the sequence read
 *  uint64_t total number of sequences
 */
void print_pos_stats(float* d_pos_stats, size_t LEN, uint64_t totalN) {
    const char charset[] = "ATGCNQ";
    
    // copy data to hist memory
    float* tmp_pos_stats = (float*)calloc(6*LEN, sizeof(float));
    CudaSafeCall( hipMemcpy(tmp_pos_stats, d_pos_stats, 6*LEN*sizeof(float), hipMemcpyDeviceToHost) );
    
    for (int b = 0; b < 6; b++) {
        printf("@%c", charset[b]);
        for (size_t i = 0; i < LEN; i++) {
            if (b < 5)
                // average nucleotide content
                printf("\t%0.4f",(float)tmp_pos_stats[6*i+b]/totalN);
            else {
                // check if q-stats are 0 at this position
                // can happen if the LEN is bigger than the actual read length
                if (tmp_pos_stats[6*i+b] == 0)
                     printf("\t%0.4f", (float)0);
                else
                    // average quality
                    printf("\t%0.4f",(float)(tmp_pos_stats[6*i+b] - 33*totalN)/totalN);
            }
        }
        printf("\n");
    }
    // cleanup
    free(tmp_pos_stats);
}

/* read_fastq attempts to read a fastq record from a file
 * Arguments:
 *  char* to hold the read data
 *  FILE* to an open file
 *  size_t for sequence read length
 *  size_t offset within the target string
 */
bool read_fastq(char* str, FILE* fp, const size_t L, size_t offset) {
    char* line = NULL;
    size_t len = 0;
    ssize_t read, rs;
    
    while (1) {
        // find entry point
        read = getline(&line, &len, fp);
        // EOF or other error
        if (read == -1)
            break;
        
        // entry point, line starts with '@'
        if (line[0] != '@') { 
            free(line); 
            continue;   // drop this line keep going
        }
            
        // read sequence string
        free(line);
        line = NULL;
        len = 0;
        read = getline(&line, &len, fp);
        // check for error and empty line
        if ((read != -1) && (read != 1)) {
            rs = read - 1;      // remove newline
            if (rs < L) 
                memcpy(str+2*L*offset,line,rs*sizeof(char)); 
            else
                memcpy(str+2*L*offset,line,L*sizeof(char));
        } else
            break;
            
        // read '+' string, line starts with '+'
        free(line);
        line = NULL;
        len = 0;
        read = getline(&line, &len, fp);
        // check for error and not starting with '+'
        if ((read != -1) && (line[0] != '+')) 
            break; 
            
        // read q-string
        free(line);
        line = NULL;
        len = 0;
        read = getline(&line, &len, fp);
        // check for error and different length from sequence
        if ((read != -1) && (read - 1 == rs)) {
            // newline eliminated already
            if (rs < L)
                memcpy(str+2*L*offset+L,line,rs*sizeof(char));
            else 
                memcpy(str+2*L*offset+L,line,L*sizeof(char));
            
            free(line);
            return true;    // success
        
        } else
            break;
    }
    
    // something went wrong
    free(line);
    return false;
}

/* main 
 * Arguments:
 *  filename
 *  sequence read length
 *  (optional) max number of sequences to process
 */
int main(int argc, char** argv) {
    // change this if needed (and change the kernel launch parameters below)
    size_t CHUNK_SZ = 1000000;
    
    if (argc < 3) {
        fprintf(stderr, "%s: Insufficient arguments\n", argv[0]);
        exit(1);
    }
    
    char* fname = argv[1];              // input filename
    size_t LEN = (size_t)atoi(argv[2]); // desired sequence length
    uint64_t maxSeq = 0;                // maximum number of sequences to process
    uint64_t totalN = 0;                // total number of sequences
    
    if (argc == 4) {
        maxSeq = (uint64_t)atoi(argv[3]);
    }
    
    // open file for reading
    FILE* fp = fopen(fname, "r");
    if (fp == NULL) { 
        err(1, "fopen: %s", fname);
        exit(2);
    }
    
    // allocate host and device memory
    char* h_str = (char*)calloc(2*LEN*CHUNK_SZ,sizeof(char));
    
    char* d_str;       // sequence
    CudaSafeCall( hipMalloc((void**)&d_str, 2*CHUNK_SZ*LEN*sizeof(char)) );
    CudaSafeCall( hipMemset(d_str, 0, 2*CHUNK_SZ*LEN*sizeof(char)) );


    float* d_pos_stats;  // pos stats    
    CudaSafeCall( hipMalloc((void**)&d_pos_stats, 6*LEN*sizeof(float)) );
    CudaSafeCall( hipMemset(d_pos_stats, 0, 6*LEN*sizeof(float)) );
    
    // main loop to go arbitrary number of sequences
    bool done = false;
    while(!done) {
        for (size_t i = 0; i < (size_t)CHUNK_SZ; i++) {
            if (read_fastq(h_str, fp, LEN, i)) {
                totalN++;
                if (totalN == maxSeq) {
                    done = true;
                    break;
                }
            
            } else { 
                done = true; 
                break;
            }
        }

        CudaSafeCall( hipMemcpy(d_str, h_str, 2*CHUNK_SZ*LEN*sizeof(char), hipMemcpyHostToDevice) );
    
        pos_stats<<<1024,1024,6*LEN*sizeof(float)>>>(d_str, d_pos_stats, LEN, CHUNK_SZ);
        CudaCheckError();
            
        // reset arrays
        memset(h_str, 0, 2*CHUNK_SZ*LEN*sizeof(char));
        CudaSafeCall( hipMemset(d_str, 0, 2*CHUNK_SZ*LEN*sizeof(char)) );
    }
    fprintf(stderr, "\n");
    
    // close file
    fclose(fp);
    
    printf("Total seqs processed: %lu\tLength: %zu\tChunk: %zu\n", totalN, LEN, CHUNK_SZ);
    print_pos_stats(d_pos_stats, LEN, totalN);
    hipDeviceSynchronize();    
    
    // cleanuup
    // device
    hipFree(d_str);
    hipFree(d_pos_stats);
    
    // host
    free(h_str);
    
    return 0;
}